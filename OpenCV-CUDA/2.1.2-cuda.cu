#include "hip/hip_runtime.h"
#include <iostream>
#include <opencv2/opencv.hpp>
#include <opencv2/cudaarithm.hpp>
#include <opencv2/cudawarping.hpp>
#include <cfloat>
#include <opencv2/core/cuda/common.hpp>
#include <opencv2/core/cuda/border_interpolate.hpp>
#include <opencv2/core/cuda/vec_traits.hpp>
#include <opencv2/core/cuda/vec_math.hpp>
#include <string>
#include <cmath>
#include <chrono>  

using namespace std;

enum AnaglyphType {
    NORMAL=0,
    TRUE,
    GRAY,
    COLOR,
    HALFCOLOR,
    OPTIMIZED
};

__global__ void generateGaussianKernelKernel(double* gaussKernel, int kernelSize, double sigma) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < kernelSize && y < kernelSize) {
        int halfKernelSize = kernelSize / 2;
        const double PI = 3.14159265358979323846;

        double lp = 1.0 / (2.0 * PI * sigma * sigma);
        double rp = 1.0 / (2.0 * sigma * sigma);

        double gaussianVal = lp * exp(-((x - halfKernelSize) * (x - halfKernelSize) + (y - halfKernelSize) * (y - halfKernelSize)) * rp);
        gaussKernel[y * kernelSize + x] = gaussianVal;
    }
}

__global__ void applyGaussianBlurKernel(const cv::cuda::PtrStepSz<uchar3> src, cv::cuda::PtrStepSz<uchar3> dst, int kernelSize, double* gaussKernel) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < src.cols && y < src.rows) {
        int halfKernelSize = kernelSize / 2;

        double sum[3] = {0.0, 0.0, 0.0};
        double gaussianTotal = 0.0;

        for (int i = -halfKernelSize; i <= halfKernelSize; ++i) {
            for (int j = -halfKernelSize; j <= halfKernelSize; ++j) {
                int row = min(max(y + i, 0), src.rows - 1);
                int col = min(max(x + j, 0), src.cols - 1);

                double gaussianVal = gaussKernel[(i + halfKernelSize) * kernelSize + (j + halfKernelSize)];
                gaussianTotal += gaussianVal;

                uchar3 pixel = src(row, col);
                double pixelVec[3] = {static_cast<double>(pixel.x), static_cast<double>(pixel.y), static_cast<double>(pixel.z)};
                for (int k = 0; k < 3; ++k) {
                    sum[k] += pixelVec[k] * gaussianVal;
                }
            }
        }

        for (int k = 0; k < 3; ++k) {
            sum[k] /= gaussianTotal;
        }
        dst(y, x) = make_uchar3(static_cast<uchar>(sum[0]), static_cast<uchar>(sum[1]), static_cast<uchar>(sum[2]));
    }
}

__global__ void processKernel(const cv::cuda::PtrStepSz<uchar3> left_image,
                                     const cv::cuda::PtrStepSz<uchar3> right_image,
                                     cv::cuda::PtrStepSz<uchar3> anaglyph_image,
                                     int anaglyph_type) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < left_image.cols && y < left_image.rows) {
        uchar3 left_pixel = left_image(y, x);
        uchar3 right_pixel = right_image(y, x);

        switch (anaglyph_type) {
            case TRUE:
                // True Anaglyphs
                anaglyph_image(y, x) = make_uchar3(
                    0.299f * right_pixel.z + 0.578f * right_pixel.y + 0.114f * right_pixel.x,
                    0,
                    0.299f * left_pixel.z + 0.578f * left_pixel.y + 0.114f * left_pixel.x
                );
                break;
            case GRAY:
                // Gray Anaglyphs
                anaglyph_image(y, x) = make_uchar3(
                    0.299f * right_pixel.x + 0.578f * right_pixel.y + 0.114f * right_pixel.z,
                    0.299f * right_pixel.x + 0.578f * right_pixel.y + 0.114f * right_pixel.z,
                    0.299f * left_pixel.x + 0.578f * left_pixel.y + 0.114f * left_pixel.z
                );
                break;
            case COLOR:
                // Color Anaglyphs
                anaglyph_image(y, x) = make_uchar3(
                    right_pixel.x,
                    right_pixel.y,
                    left_pixel.z
                );
                break;
            case HALFCOLOR:
                // Half Color Anaglyphs
                anaglyph_image(y, x) = make_uchar3(
                    0.299f * right_pixel.x + 0.578f * right_pixel.y + 0.114f * right_pixel.z,
                    right_pixel.y,
                    left_pixel.z
                );
                break;
            case OPTIMIZED:
                // Optimized Anaglyphs
                anaglyph_image(y, x) = make_uchar3(
                    0.7f * right_pixel.y + 0.3f * right_pixel.x,
                    right_pixel.y,
                    left_pixel.z
                );
                break;
            default:
                // No Anaglyphs
                anaglyph_image(y, x) = left_pixel;
        }
    }
}

int divUp(int a, int b)
{
    // const dim3 grid((d_left_image.cols + block.x - 1) / block.x, (d_left_image.rows + block.y - 1) / block.y);
    return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

void processCUDA(const cv::cuda::GpuMat& d_left_image,
                const cv::cuda::GpuMat& d_right_image,
                cv::cuda::GpuMat& d_anaglyph_image,
                int kernelSize,
                double sigma,
                int anaglyph_type,
                double* gaussKernel) {
    const dim3 block(64, 16);
    const dim3 grid(divUp(d_right_image.cols, block.x), divUp(d_right_image.rows, block.y));

    // Apply Gaussian blur kernel
    applyGaussianBlurKernel<<<grid, block>>>(d_left_image, d_left_image, kernelSize, gaussKernel);
    applyGaussianBlurKernel<<<grid, block>>>(d_right_image, d_right_image, kernelSize, gaussKernel);

    if (anaglyph_type == NORMAL) {
        d_anaglyph_image = d_left_image;
        return;
    }
    // Create anaglyph image kernel
    processKernel<<<grid, block>>>(d_left_image, d_right_image, d_anaglyph_image, anaglyph_type);
}

int main( int argc, char** argv )
{
    if (argc < 5) {
        cerr << "Usage: " << argv[0] << " <image_path> <anaglyph_type> <kernel_size> <sigma>" << endl;
        return -1;
    }

    // Read the stereo image
    cv::Mat stereo_image = cv::imread(argv[1], cv::IMREAD_COLOR);

    // Determine the type of anaglyphs to generate
    AnaglyphType anaglyph_type = static_cast<AnaglyphType>(atoi(argv[2]));

    // Check if the image is loaded successfully
    if (stereo_image.empty()) {
        cerr << "Error: Unable to load image." << endl;
        return -1;
    }

    if (anaglyph_type < NORMAL || anaglyph_type > OPTIMIZED) {
        cerr << "Error: Invalid anaglyph type." << endl;
        cerr << "Anaglyph types:" << endl;
        cerr << "0: None Anaglyphs" << endl;
        cerr << "1: True Anaglyphs" << endl;
        cerr << "2: Gray Anaglyphs" << endl;
        cerr << "3: Color Anaglyphs" << endl;
        cerr << "4: Half Color Anaglyphs" << endl;
        cerr << "5: Optimized Anaglyphs" << endl;
        return -1;
    }

    // Split the stereo image into left and right images
    cv::Mat left_image(stereo_image, cv::Rect(0, 0, stereo_image.cols / 2, stereo_image.rows));
    cv::Mat right_image(stereo_image, cv::Rect(stereo_image.cols / 2, 0, stereo_image.cols / 2, stereo_image.rows));

    int kernelSize = atoi(argv[3]);
    double sigma = atof(argv[4]);

    if (!kernelSize|| !sigma) {
        cerr << "Error: Invalid kernel size or sigma." << endl;
        cerr << "Input kernel size in range odd numbers from 3 to 21" << endl;
        cerr << "Input sigma in range odd numbers from 0.1 to 10" << endl;
        return -1;
    }

    double* gaussKernel;
    hipMalloc(&gaussKernel, kernelSize * kernelSize * sizeof(double));

    // Initialize the memory to 0
    hipMemset(gaussKernel, 0, kernelSize * kernelSize * sizeof(double));

    dim3 blockSize(16, 16); // You can adjust these values as needed
    dim3 gridSize((kernelSize + blockSize.x - 1) / blockSize.x, (kernelSize + blockSize.y - 1) / blockSize.y);

    generateGaussianKernelKernel<<<gridSize, blockSize>>>(gaussKernel, kernelSize, sigma);

    cv::Mat anaglyph_image(left_image.size(), CV_8UC3);

    std::string anaglyph_name;
    switch (anaglyph_type) {
        case TRUE:
            anaglyph_name = "True";
            break;
        case GRAY:
            anaglyph_name = "Gray";
            break;
        case COLOR:
            anaglyph_name = "Color";
            break;
        case HALFCOLOR:
            anaglyph_name = "Half Color";
            break;
        case OPTIMIZED:
            anaglyph_name = "Optimized";
            break;
        default:
            anaglyph_name = "None";
    }

    // Convert input images to GPU Mat
    cv::cuda::GpuMat d_left_image, d_right_image, d_anaglyph_image;
    d_left_image.upload(left_image);
    d_right_image.upload(right_image);

    // Allocate memory for the anaglyph image on GPU
    d_anaglyph_image.create(left_image.rows, left_image.cols, CV_8UC3);

    // Start the timer
    auto begin = chrono::high_resolution_clock::now();

    // Number of iterations
    const int iter = 1;

    // Perform the operation iter times
    for (int it = 0; it < iter; it++) {
        processCUDA(d_left_image, d_right_image, d_anaglyph_image, kernelSize, sigma, anaglyph_type, gaussKernel);
    }

    // Stop the timer
    auto end = chrono::high_resolution_clock::now();

    // Download the result from the GPU
    d_anaglyph_image.download(anaglyph_image);

    // Calculate the time difference
    chrono::duration<double> diff = end - begin;

    // Display the original images
    cv::imshow("Input Image", stereo_image);

    // Display the output image
    cv::imshow("Gaussian +" + anaglyph_name + " Anaglyph Image", anaglyph_image);

    // Save the anaglyph image
    // std::string filename =  "output/2.1.2/" + anaglyph_name + "Anaglyph-blurred.jpg";
    // cv::imwrite(filename, anaglyph_image);

    // Display performance metrics
    cout << "Total time: " << diff.count() << " s" << endl;
    cout << "Time for 1 iteration: " << diff.count() / iter << " s" << endl;
    cout << "IPS: " << iter / diff.count() << endl;

    // Wait for a key press before closing the windows
    cv::waitKey();

    hipFree(gaussKernel);

    return 0;
}
