#include "hip/hip_runtime.h"
#include <iostream>
#include <opencv2/opencv.hpp>
#include <string>
#include <cmath>
#include <chrono>  // for high_resolution_clock
#include <hip/hip_runtime.h>

using namespace std;

__global__ void calculateAndDenoiseKernel(const uchar3* src, uchar3* dst, int cols, int rows, int neighborhoodSize, float factorRatio) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < cols && y < rows) {
        int halfSize = neighborhoodSize / 2;
        int xStart = max(0, x - halfSize);
        int yStart = max(0, y - halfSize);
        int xEnd = min(cols, x + halfSize);
        int yEnd = min(rows, y + halfSize);

        float3 mean = make_float3(0.0f, 0.0f, 0.0f);
        float3 cov = make_float3(0.0f, 0.0f, 0.0f);

        for (int j = yStart; j < yEnd; ++j) {
            for (int i = xStart; i < xEnd; ++i) {
                uchar3 pixel = src[j * cols + i];
                mean.x += pixel.x;
                mean.y += pixel.y;
                mean.z += pixel.z;
            }
        }

        int count = (xEnd - xStart) * (yEnd - yStart);
        mean.x /= count;
        mean.y /= count;
        mean.z /= count;

        for (int j = yStart; j < yEnd; ++j) {
            for (int i = xStart; i < xEnd; ++i) {
                uchar3 pixel = src[j * cols + i];
                float3 diff = make_float3(pixel.x - mean.x, pixel.y - mean.y, pixel.z - mean.z);
                cov.x += diff.x * diff.x;
                cov.y += diff.y * diff.y;
                cov.z += diff.z * diff.z;
            }
        }

        cov.x /= count;
        cov.y /= count;
        cov.z /= count;

        float determinant = cov.x * cov.y * cov.z;

        int kernelSize;
        if (determinant != 0) {
            kernelSize = static_cast<int>(round(factorRatio / determinant));
            kernelSize = kernelSize % 2 == 0 ? kernelSize + 1 : kernelSize;
        } else {
            kernelSize = neighborhoodSize;
        }

        kernelSize = max(1, kernelSize);
        kernelSize |= 1; // Ensure it's odd

        float3 sum = make_float3(0.0f, 0.0f, 0.0f);
        count = 0;

        for (int j = y - kernelSize / 2; j <= y + kernelSize / 2; ++j) {
            for (int i = x - kernelSize / 2; i <= x + kernelSize / 2; ++i) {
                if (i >= 0 && i < cols && j >= 0 && j < rows) {
                    uchar3 pixel = src[j * cols + i];
                    sum.x += pixel.x;
                    sum.y += pixel.y;
                    sum.z += pixel.z;
                    ++count;
                }
            }
        }

        sum.x /= count;
        sum.y /= count;
        sum.z /= count;

        dst[y * cols + x] = make_uchar3(static_cast<unsigned char>(sum.x), static_cast<unsigned char>(sum.y), static_cast<unsigned char>(sum.z));
    }
}

void processCUDA(const cv::cuda::GpuMat& src, cv::cuda::GpuMat& dst, int neighborhoodSize, double factorRatio) {
    dim3 block(32, 8);
    dim3 grid((src.cols + block.x - 1) / block.x, (src.rows + block.y - 1) / block.y);

    calculateAndDenoiseKernel<<<grid, block>>>(
        reinterpret_cast<uchar3*>(const_cast<unsigned char*>(src.ptr())), 
        reinterpret_cast<uchar3*>(dst.ptr()), 
        src.cols, src.rows, neighborhoodSize, factorRatio);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        cerr << "CUDA error in calculateAndDenoiseKernel: " << hipGetErrorString(err) << endl;
        return;
    }
}

int main(int argc, char** argv) {
    if (argc < 4) {
        cerr << "Usage: " << argv[0] << " <image_path> <neighborhood_size> <factor_ratio>" << endl;
        return -1;
    }

    cv::Mat stereo_image = cv::imread(argv[1], cv::IMREAD_COLOR);
    if (stereo_image.empty()) {
        cerr << "Error: Unable to load image." << endl;
        return -1;
    }

    int neighborhoodSize = atoi(argv[2]);
    double factorRatio = atof(argv[3]);

    if (!factorRatio || !neighborhoodSize) {
        cerr << "Error: Invalid input." << endl;
        cerr << "Error: Neighborhood size must be an odd number." << endl;
        cerr << "Error: Factor ratio must be greater than 0." << endl;
        return -1;
    }
    
    if (neighborhoodSize % 2 == 0) {
        cerr << "Error: Neighborhood size must be an odd number." << endl;
        return -1;
    }

    if (factorRatio <= 0) {
        cerr << "Error: Factor ratio must be greater than 0." << endl;
        return -1;
    }

    cv::cuda::GpuMat d_stereo_image;
    d_stereo_image.upload(stereo_image);

    cv::cuda::GpuMat d_denoised_image(d_stereo_image.size(), d_stereo_image.type());

    // Start the timer
    auto begin = chrono::high_resolution_clock::now();

    // Number of iterations
    const int iter = 5;

    for (int it = 0; it < iter; it++) {
        processCUDA(d_stereo_image, d_denoised_image, neighborhoodSize, factorRatio);
    }

    // Stop the timer
    auto end = std::chrono::high_resolution_clock::now();

    // Calculate the time difference
    chrono::duration<double> diff = end - begin;

    // Download the result back to CPU
    cv::Mat denoised_image(stereo_image.size(), stereo_image.type());
    d_denoised_image.download(denoised_image);

    // Display performance metrics
    cout << "Total time: " << diff.count() << " s" << endl;
    cout << "Time for 1 iteration: " << diff.count() / iter << " s" << endl;
    cout << "IPS: " << iter / diff.count() << endl;

    // Display the original and processed images
    cv::imshow("Original Image", stereo_image);
    cv::imshow("Denoised Image", denoised_image);
    cv::waitKey();

    d_stereo_image.release();
    d_denoised_image.release();

    return 0;
}
